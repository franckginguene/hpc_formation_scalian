
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10  // Nombre de donn�es � traiter

// Macro utilitaire de retour d'erreur	
#define CUT_CHECK_ERROR(errorMessage) {											\
	  hipError_t err = hipGetLastError();										\
	  if( hipSuccess != err) {													\
		  fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",		\
				  errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );	\
		  exit(EXIT_FAILURE);													\
	  }																			\
  }

//
// Noyau CUDA
//
__global__ void Reverse (int *dataIn, int *dataOut)
{
	// TODO : boucle for
}

//
// Code du Host
//
int main (void)
{  
    int dataIn[N];     // Donn�es � traiter (CPU)
    int dataOut[N];    // Donn�es r�sultats (CPU)
    int *dev_dataIn;   // Donn�es � traiter (GPU)
    int *dev_dataOut;  // Donn�es r�sultats (GPU)

    // Allocation des vecteurs sur le device
    // TODO : allouer les 2 tableaux dev_dataIn et dev_dataOut de taille N

    // Initialisation des donn�es
    printf ("Data In:  ");
    for (int i = 0; i < N; i++) {
        dataIn[i] = i;
        printf ("%d ", dataIn[i]);
    }
    printf ("\n");

    // Copie des donn�es �  traiter sur le GPU.
	// TODO : copier les donn�es de dataIn vers dev_dataIn avec cudaMemcpy

    // Lancement du noyau.
    Reverse<<<1, 1>>>( dev_dataIn, dev_dataOut );
    CUT_CHECK_ERROR("Kernel Execution Failed!");

    // Copie des donn�es r�sultats du GPU vers le host.
	// TODO : copie de dev_dataOut vers dataOut

    // Affichage du r�sultat
    printf ("Data Out: ");
    for (int i = 0; i < N; i++)
        printf ("%d ", dataOut[i]);
    printf ("\n");

    // Lib�ration des vecteurs sur le device
	// TODO lib�ration de la m�moire

    return 0 ;
}
